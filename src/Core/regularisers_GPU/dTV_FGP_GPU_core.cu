#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2017 Daniil Kazantsev
Copyright 2017 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "shared.h"
#include "dTV_FGP_GPU_core.h"
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

/* CUDA implementation of FGP-dTV [1,2] denoising/regularization model (2D/3D case)
 * which employs structural similarity of the level sets of two images/volumes, see [1,2]
 * The current implementation updates image 1 while image 2 is being fixed.
 *
 * Input Parameters:
 * 1. Noisy image/volume [REQUIRED]
 * 2. Additional reference image/volume of the same dimensions as (1) [REQUIRED]
 * 3. lambdaPar - regularization parameter [REQUIRED]
 * 4. Number of iterations [OPTIONAL]
 * 5. eplsilon: tolerance constant [OPTIONAL]
 * 6. eta: smoothing constant to calculate gradient of the reference [OPTIONAL] *
 * 7. TV-type: methodTV - 'iso' (0) or 'l1' (1) [OPTIONAL]
 * 8. nonneg: 'nonnegativity (0 is OFF by default) [OPTIONAL]
 * 9. GPU device number if for multigpu run (default 0) [OPTIONAL]

 * Output:
 * [1] Filtered/regularized image/volume
 * [2] Information vector which contains [iteration no., reached tolerance]
 *
 * This function is based on the Matlab's codes and papers by
 * [1] Amir Beck and Marc Teboulle, "Fast Gradient-Based Algorithms for Constrained Total Variation Image Denoising and Deblurring Problems"
 * [2] M. J. Ehrhardt and M. M. Betcke, Multi-Contrast MRI Reconstruction with Structure-Guided Total Variation, SIAM Journal on Imaging Sciences 9(3), pp. 1084–1106
 */


#define BLKXSIZE2D 16
#define BLKYSIZE2D 16

#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8

#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )
//struct square { __host__ __device__ float operator()(float x) { return x * x; } };

/************************************************/
/*****************2D modules*********************/
/************************************************/

__global__ void GradNorm_func2D_kernel(float *Refd, float *Refd_x, float *Refd_y, float eta, int N, int M, int ImSize)
{

    float val1, val2, gradX, gradY, magn;
    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {
        /* boundary conditions */
        if (xIndex >= N-1) val1 = 0.0f; else val1 =  Refd[(xIndex+1) + N*yIndex];
        if (yIndex >= M-1) val2 = 0.0f; else val2 =  Refd[(xIndex) + N*(yIndex + 1)];

            gradX = val1 - Refd[index];
            gradY = val2 - Refd[index];
            magn = pow(gradX,2) + pow(gradY,2);
            magn = sqrt(magn + pow(eta,2));
            Refd_x[index] = gradX/magn;
            Refd_y[index] = gradY/magn;
    }
    return;
}

__global__ void ProjectVect_func2D_kernel(float *R1, float *R2, float *Refd_x, float *Refd_y, int N, int M, int ImSize)
{

    float in_prod;
    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {
        in_prod = R1[index]*Refd_x[index] + R2[index]*Refd_y[index];   /* calculate inner product */
        R1[index] = R1[index] - in_prod*Refd_x[index];
        R2[index] = R2[index] - in_prod*Refd_y[index];
    }
    return;
}


__global__ void Obj_dfunc2D_kernel(float *Ad, float *D, float *R1, float *R2, int N, int M, int ImSize, float lambda)
{

    float val1,val2;

    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {
        if (xIndex <= 0) {val1 = 0.0f;} else {val1 = R1[(xIndex-1) + N*yIndex];}
        if (yIndex <= 0) {val2 = 0.0f;} else {val2 = R2[xIndex + N*(yIndex-1)];}

        //Write final result to global memory
        D[index] = Ad[index] - lambda*(R1[index] + R2[index] - val1 - val2);
    }
    return;
}

__global__ void Grad_dfunc2D_kernel(float *P1, float *P2, float *D, float *R1, float *R2,  float *Refd_x, float *Refd_y, int N, int M, int ImSize, float multip)
{

    float val1,val2,in_prod;

    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {

        /* boundary conditions */
        if (xIndex >= N-1) val1 = 0.0f; else val1 = D[index] - D[(xIndex+1) + N*yIndex];
        if (yIndex >= M-1) val2 = 0.0f; else val2 = D[index] - D[(xIndex) + N*(yIndex + 1)];

        in_prod = val1*Refd_x[index] + val2*Refd_y[index];   /* calculate inner product */
        val1 = val1 - in_prod*Refd_x[index];
        val2 = val2 - in_prod*Refd_y[index];

        //Write final result to global memory
        P1[index] = R1[index] + multip*val1;
        P2[index] = R2[index] + multip*val2;
    }
    return;
}

__global__ void Proj_dfunc2D_iso_kernel(float *P1, float *P2, int N, int M, int ImSize)
{

    float denom;
    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {
        denom = pow(P1[index],2) +  pow(P2[index],2);
        if (denom > 1.0f) {
            P1[index] = P1[index]/sqrt(denom);
            P2[index] = P2[index]/sqrt(denom);
        }
    }
    return;
}
__global__ void Proj_dfunc2D_aniso_kernel(float *P1, float *P2, int N, int M, int ImSize)
{

    float val1, val2;
    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {
                val1 = abs(P1[index]);
                val2 = abs(P2[index]);
                if (val1 < 1.0f) {val1 = 1.0f;}
                if (val2 < 1.0f) {val2 = 1.0f;}
                P1[index] = P1[index]/val1;
                P2[index] = P2[index]/val2;
    }
    return;
}
__global__ void Rupd_dfunc2D_kernel(float *P1, float *P1_old, float *P2, float *P2_old, float *R1, float *R2, float tkp1, float tk, float multip2, int N, int M, int ImSize)
{
    //calculate each thread global index
    const int xIndex=blockIdx.x*blockDim.x+threadIdx.x;
    const int yIndex=blockIdx.y*blockDim.y+threadIdx.y;

    int index = xIndex + N*yIndex;

    if ((xIndex < N) && (yIndex < M)) {
        R1[index] = P1[index] + multip2*(P1[index] - P1_old[index]);
        R2[index] = P2[index] + multip2*(P2[index] - P2_old[index]);
    }
    return;
}
__global__ void dTVnonneg2D_kernel(float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        if (Output[index] < 0.0f) Output[index] = 0.0f;
    }
}
__global__ void dTVcopy_kernel2D(float *Input, float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        Output[index] = Input[index];
    }
}

__global__ void dTVcopy_kernel3D(float *Input, float* Output, int N, int M, int Z, int num_total)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        Output[index] = Input[index];
    }
}

__global__ void dTVResidCalc2D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        Output[index] = Input1[index] - Input2[index];
    }
}

__global__ void dTVResidCalc3D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int Z, int num_total)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        Output[index] = Input1[index] - Input2[index];
    }
}

/************************************************/
/*****************3D modules*********************/
/************************************************/
__global__ void GradNorm_func3D_kernel(float *Refd, float *Refd_x, float *Refd_y, float *Refd_z, float eta, int N, int M, int Z, int ImSize)
{

    float val1, val2, val3, gradX, gradY, gradZ, magn;
    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k < Z)) {
        /* boundary conditions */
        if (i >= N-1) val1 = 0.0f; else val1 =  Refd[(N*M)*k + (i+1) + N*j];
        if (j >= M-1) val2 = 0.0f; else val2 =  Refd[(N*M)*k + i + N*(j+1)];
        if (k >= Z-1) val3 = 0.0f; else val3 =  Refd[(N*M)*(k+1) + i + N*j];

            gradX = val1 - Refd[index];
            gradY = val2 - Refd[index];
            gradZ = val3 - Refd[index];
            magn = pow(gradX,2) + pow(gradY,2) + pow(gradZ,2);
            magn = sqrt(magn + pow(eta,2));
            Refd_x[index] = gradX/magn;
            Refd_y[index] = gradY/magn;
            Refd_z[index] = gradZ/magn;
    }
    return;
}

__global__ void ProjectVect_func3D_kernel(float *R1, float *R2, float *R3, float *Refd_x, float *Refd_y, float *Refd_z, int N, int M, int Z, int ImSize)
{

    float in_prod;
    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k < Z)) {
        in_prod = R1[index]*Refd_x[index] + R2[index]*Refd_y[index] + R3[index]*Refd_z[index]; /* calculate inner product */

        R1[index] = R1[index] - in_prod*Refd_x[index];
        R2[index] = R2[index] - in_prod*Refd_y[index];
        R3[index] = R3[index] - in_prod*Refd_z[index];
    }
    return;
}


__global__ void Obj_dfunc3D_kernel(float *Ad, float *D, float *R1, float *R2, float *R3, int N, int M, int Z, int ImSize, float lambda)
{

    float val1,val2,val3;

    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k < Z)) {
        if (i <= 0) {val1 = 0.0f;} else {val1 = R1[(N*M)*(k) + (i-1) + N*j];}
        if (j <= 0) {val2 = 0.0f;} else {val2 = R2[(N*M)*(k) + i + N*(j-1)];}
        if (k <= 0) {val3 = 0.0f;} else {val3 = R3[(N*M)*(k-1) + i + N*j];}
        //Write final result to global memory
        D[index] = Ad[index] - lambda*(R1[index] + R2[index] + R3[index] - val1 - val2 - val3);
    }
    return;
}

__global__ void Grad_dfunc3D_kernel(float *P1, float *P2, float *P3, float *D, float *R1, float *R2, float *R3, float *Refd_x, float *Refd_y, float *Refd_z, int N, int M, int Z, int ImSize, float multip)
{

    float val1,val2,val3,in_prod;

    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k <  Z)) {
        /* boundary conditions */
        if (i >= N-1) val1 = 0.0f; else val1 = D[index] - D[(N*M)*(k) + (i+1) + N*j];
        if (j >= M-1) val2 = 0.0f; else val2 = D[index] - D[(N*M)*(k) + i + N*(j+1)];
        if (k >= Z-1) val3 = 0.0f; else val3 = D[index] - D[(N*M)*(k+1) + i + N*j];

        in_prod = val1*Refd_x[index] + val2*Refd_y[index] + val3*Refd_z[index];   /* calculate inner product */
        val1 = val1 - in_prod*Refd_x[index];
        val2 = val2 - in_prod*Refd_y[index];
        val3 = val3 - in_prod*Refd_z[index];

        //Write final result to global memory
        P1[index] = R1[index] + multip*val1;
        P2[index] = R2[index] + multip*val2;
        P3[index] = R3[index] + multip*val3;
    }
    return;
}

__global__ void Proj_dfunc3D_iso_kernel(float *P1, float *P2, float *P3, int N, int M, int Z, int ImSize)
{

    float denom,sq_denom;
    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k <  Z)) {
        denom = pow(P1[index],2) +  pow(P2[index],2) + pow(P3[index],2);

        if (denom > 1.0f) {
            sq_denom = 1.0f/sqrt(denom);
            P1[index] = P1[index]*sq_denom;
            P2[index] = P2[index]*sq_denom;
            P3[index] = P3[index]*sq_denom;
        }
    }
    return;
}

__global__ void Proj_dfunc3D_aniso_kernel(float *P1, float *P2, float *P3, int N, int M, int Z, int ImSize)
{

    float val1, val2, val3;
    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k <  Z)) {
                val1 = abs(P1[index]);
                val2 = abs(P2[index]);
                val3 = abs(P3[index]);
                if (val1 < 1.0f) {val1 = 1.0f;}
                if (val2 < 1.0f) {val2 = 1.0f;}
                if (val3 < 1.0f) {val3 = 1.0f;}
                P1[index] = P1[index]/val1;
                P2[index] = P2[index]/val2;
                P3[index] = P3[index]/val3;
    }
    return;
}


__global__ void Rupd_dfunc3D_kernel(float *P1, float *P1_old, float *P2, float *P2_old, float *P3, float *P3_old, float *R1, float *R2, float *R3, float tkp1, float tk, float multip2, int N, int M, int Z, int ImSize)
{
    //calculate each thread global index
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if ((i < N) && (j < M) && (k <  Z)) {
        R1[index] = P1[index] + multip2*(P1[index] - P1_old[index]);
        R2[index] = P2[index] + multip2*(P2[index] - P2_old[index]);
        R3[index] = P3[index] + multip2*(P3[index] - P3_old[index]);
    }
    return;
}

__global__ void dTVnonneg3D_kernel(float* Output, int N, int M, int Z, int num_total)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        if (Output[index] < 0.0f) Output[index] = 0.0f;
    }
}
/*%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%*/

////////////MAIN HOST FUNCTION ///////////////
extern "C" int dTV_FGP_GPU_main(float *Input, float *InputRef, float *Output, float *infovector, float lambdaPar, int iter, float epsil, float eta, int methodTV, int nonneg, int gpu_device, int dimX, int dimY, int dimZ)
{
    int deviceCount = -1; // number of devices
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        return -1;
    }

    checkCudaErrors(hipSetDevice(gpu_device)); 

    int count = 0, i;
    float re, multip,multip2;
    re = 0.0f;
	  float tk = 1.0f;
    float tkp1=1.0f;

    if (dimZ <= 1) {
		/*2D verson*/
		int ImSize = dimX*dimY;
		float *d_input, *d_update=NULL, *d_update_prev=NULL, *P1=NULL, *P2=NULL, *P1_prev=NULL, *P2_prev=NULL, *R1=NULL, *R2=NULL, *InputRef_x=NULL, *InputRef_y=NULL, *d_InputRef=NULL;

		dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
		dim3 dimGrid(idivup(dimX,BLKXSIZE2D), idivup(dimY,BLKYSIZE2D));

		/*allocate space for images on device*/
		checkCudaErrors( hipMalloc((void**)&d_input,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&d_update,ImSize*sizeof(float)) );
		if (epsil != 0.0f) checkCudaErrors( hipMalloc((void**)&d_update_prev,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&P1,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&P2,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&P1_prev,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&P2_prev,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&R1,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&R2,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&d_InputRef,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&InputRef_x,ImSize*sizeof(float)) );
		checkCudaErrors( hipMalloc((void**)&InputRef_y,ImSize*sizeof(float)) );

        checkCudaErrors( hipMemcpy(d_input,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));
        checkCudaErrors( hipMemcpy(d_InputRef,InputRef,ImSize*sizeof(float),hipMemcpyHostToDevice));

        hipMemset(P1, 0, ImSize*sizeof(float));
        hipMemset(P2, 0, ImSize*sizeof(float));
        hipMemset(P1_prev, 0, ImSize*sizeof(float));
        hipMemset(P2_prev, 0, ImSize*sizeof(float));
        hipMemset(R1, 0, ImSize*sizeof(float));
        hipMemset(R2, 0, ImSize*sizeof(float));
        hipMemset(InputRef_x, 0, ImSize*sizeof(float));
        hipMemset(InputRef_y, 0, ImSize*sizeof(float));

        /******************** Run CUDA 2D kernel here ********************/
        multip = (1.0f/(8.0f*lambdaPar));
        /* calculate gradient vectors for the reference */
        GradNorm_func2D_kernel<<<dimGrid,dimBlock>>>(d_InputRef, InputRef_x, InputRef_y, eta, dimX, dimY, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );

        /* The main kernel */
        for (i = 0; i < iter; i++) {

            if ((epsil != 0.0f) && (i % 5 == 0)) {
              dTVcopy_kernel2D<<<dimGrid,dimBlock>>>(d_update, d_update_prev, dimX, dimY, ImSize);
              checkCudaErrors( hipDeviceSynchronize() );
              checkCudaErrors(hipPeekAtLastError() );
            }

            /*projects a 2D vector field R-1,2 onto the orthogonal complement of another 2D vector field InputRef_xy*/
            ProjectVect_func2D_kernel<<<dimGrid,dimBlock>>>(R1, R2, InputRef_x, InputRef_y, dimX, dimY, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            /* computing the gradient of the objective function */
            Obj_dfunc2D_kernel<<<dimGrid,dimBlock>>>(d_input, d_update, R1, R2, dimX, dimY, ImSize, lambdaPar);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            if (nonneg != 0) {
            dTVnonneg2D_kernel<<<dimGrid,dimBlock>>>(d_update, dimX, dimY, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() ); }

            /*Taking a step towards minus of the gradient*/
            Grad_dfunc2D_kernel<<<dimGrid,dimBlock>>>(P1, P2, d_update, R1, R2, InputRef_x, InputRef_y, dimX, dimY, ImSize, multip);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            /* projection step */
            if (methodTV == 0) Proj_dfunc2D_iso_kernel<<<dimGrid,dimBlock>>>(P1, P2, dimX, dimY, ImSize); /*isotropic TV*/
            else Proj_dfunc2D_aniso_kernel<<<dimGrid,dimBlock>>>(P1, P2, dimX, dimY, ImSize); /*anisotropic TV*/
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            tkp1 = (1.0f + sqrt(1.0f + 4.0f*tk*tk))*0.5f;
            multip2 = ((tk-1.0f)/tkp1);

            Rupd_dfunc2D_kernel<<<dimGrid,dimBlock>>>(P1, P1_prev, P2, P2_prev, R1, R2, tkp1, tk, multip2, dimX, dimY, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            dTVcopy_kernel2D<<<dimGrid,dimBlock>>>(P1, P1_prev, dimX, dimY, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            dTVcopy_kernel2D<<<dimGrid,dimBlock>>>(P2, P2_prev, dimX, dimY, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            tk = tkp1;

            if ((epsil != 0.0f) && (i % 5 == 0)) {
                /* calculate norm - stopping rules using the Thrust library */
                dTVResidCalc2D_kernel<<<dimGrid,dimBlock>>>(d_update, d_update_prev, P1, dimX, dimY, ImSize);
                checkCudaErrors( hipDeviceSynchronize() );
                checkCudaErrors(hipPeekAtLastError() );

                // setup arguments
		            square<float>        unary_op;
		            thrust::plus<float> binary_op;
                thrust::device_vector<float> d_vec(P1, P1 + ImSize);
		            float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
                thrust::device_vector<float> d_vec2(d_update, d_update + ImSize);
      		      float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

                // compute norm
                re = (reduction/reduction2);
                if (re < epsil)  count++;
                if (count > 3) break;
            }

        }
            /***************************************************************/
            //copy result matrix from device to host memory
            hipMemcpy(Output,d_update,ImSize*sizeof(float),hipMemcpyDeviceToHost);

            hipFree(d_input);
            hipFree(d_update);
            if (epsil != 0.0f) hipFree(d_update_prev);
            hipFree(P1);
            hipFree(P2);
            hipFree(P1_prev);
            hipFree(P2_prev);
            hipFree(R1);
            hipFree(R2);

            hipFree(d_InputRef);
            hipFree(InputRef_x);
            hipFree(InputRef_y);
    }
    else {
            /*3D verson*/
            int ImSize = dimX*dimY*dimZ;
            float *d_input, *d_update=NULL, *d_update_prev, *P1=NULL, *P2=NULL, *P3=NULL, *P1_prev=NULL, *P2_prev=NULL, *P3_prev=NULL, *R1=NULL, *R2=NULL, *R3=NULL, *InputRef_x=NULL, *InputRef_y=NULL, *InputRef_z=NULL, *d_InputRef=NULL;

            dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
            dim3 dimGrid(idivup(dimX,BLKXSIZE), idivup(dimY,BLKYSIZE),idivup(dimZ,BLKZSIZE));

            /*allocate space for images on device*/
            checkCudaErrors( hipMalloc((void**)&d_input,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&d_update,ImSize*sizeof(float)) );
            if (epsil != 0.0f) checkCudaErrors( hipMalloc((void**)&d_update_prev,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&P1,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&P2,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&P3,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&P1_prev,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&P2_prev,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&P3_prev,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&R1,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&R2,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&R3,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&d_InputRef,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&InputRef_x,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&InputRef_y,ImSize*sizeof(float)) );
            checkCudaErrors( hipMalloc((void**)&InputRef_z,ImSize*sizeof(float)) );

            checkCudaErrors( hipMemcpy(d_input,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors( hipMemcpy(d_InputRef,InputRef,ImSize*sizeof(float),hipMemcpyHostToDevice));

            hipMemset(P1, 0, ImSize*sizeof(float));
            hipMemset(P2, 0, ImSize*sizeof(float));
            hipMemset(P3, 0, ImSize*sizeof(float));
            hipMemset(P1_prev, 0, ImSize*sizeof(float));
            hipMemset(P2_prev, 0, ImSize*sizeof(float));
            hipMemset(P3_prev, 0, ImSize*sizeof(float));
            hipMemset(R1, 0, ImSize*sizeof(float));
            hipMemset(R2, 0, ImSize*sizeof(float));
            hipMemset(R3, 0, ImSize*sizeof(float));
            hipMemset(InputRef_x, 0, ImSize*sizeof(float));
            hipMemset(InputRef_y, 0, ImSize*sizeof(float));
            hipMemset(InputRef_z, 0, ImSize*sizeof(float));

            /********************** Run CUDA 3D kernel here ********************/
            multip = (1.0f/(26.0f*lambdaPar));
            /* calculate gradient vectors for the reference */
            GradNorm_func3D_kernel<<<dimGrid,dimBlock>>>(d_InputRef, InputRef_x, InputRef_y, InputRef_z, eta, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            /* The main kernel */
        for (i = 0; i < iter; i++) {

            if ((epsil != 0.0f) && (i % 5 == 0)) {
            dTVcopy_kernel3D<<<dimGrid,dimBlock>>>(d_update, d_update_prev, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );
            }

			      /*projects a 3D vector field R-1,2,3 onto the orthogonal complement of another 3D vector field InputRef_xyz*/
            ProjectVect_func3D_kernel<<<dimGrid,dimBlock>>>(R1, R2, R3, InputRef_x, InputRef_y, InputRef_z, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            /* computing the gradient of the objective function */
            Obj_dfunc3D_kernel<<<dimGrid,dimBlock>>>(d_input, d_update, R1, R2, R3, dimX, dimY, dimZ, ImSize, lambdaPar);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            if (nonneg != 0) {
            dTVnonneg3D_kernel<<<dimGrid,dimBlock>>>(d_update, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() ); }

            /*Taking a step towards minus of the gradient*/
            Grad_dfunc3D_kernel<<<dimGrid,dimBlock>>>(P1, P2, P3, d_update, R1, R2, R3, InputRef_x, InputRef_y, InputRef_z, dimX, dimY, dimZ, ImSize, multip);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            /* projection step */
            if (methodTV == 0) Proj_dfunc3D_iso_kernel<<<dimGrid,dimBlock>>>(P1, P2, P3, dimX, dimY, dimZ, ImSize); /* isotropic kernel */
            else Proj_dfunc3D_aniso_kernel<<<dimGrid,dimBlock>>>(P1, P2, P3, dimX, dimY, dimZ, ImSize); /* anisotropic kernel */
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            tkp1 = (1.0f + sqrt(1.0f + 4.0f*tk*tk))*0.5f;
            multip2 = ((tk-1.0f)/tkp1);

            Rupd_dfunc3D_kernel<<<dimGrid,dimBlock>>>(P1, P1_prev, P2, P2_prev, P3, P3_prev, R1, R2, R3, tkp1, tk, multip2, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            dTVcopy_kernel3D<<<dimGrid,dimBlock>>>(P1, P1_prev, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            dTVcopy_kernel3D<<<dimGrid,dimBlock>>>(P2, P2_prev, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            dTVcopy_kernel3D<<<dimGrid,dimBlock>>>(P3, P3_prev, dimX, dimY, dimZ, ImSize);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors(hipPeekAtLastError() );

            tk = tkp1;
            if ((epsil != 0.0f) && (i % 5 == 0)) {
                /* calculate norm - stopping rules using the Thrust library */
                dTVResidCalc3D_kernel<<<dimGrid,dimBlock>>>(d_update, d_update_prev, P1, dimX, dimY, dimZ, ImSize);
                checkCudaErrors( hipDeviceSynchronize() );
                checkCudaErrors(hipPeekAtLastError() );

                // setup arguments
		            square<float>        unary_op;
		            thrust::plus<float> binary_op;
                thrust::device_vector<float> d_vec(P1, P1 + ImSize);
		            float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
                thrust::device_vector<float> d_vec2(d_update, d_update + ImSize);
      		      float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

                // compute norm
                re = (reduction/reduction2);
                if (re < epsil)  count++;
                if (count > 3) break;
            }
        }
            /***************************************************************/
            //copy result matrix from device to host memory
            hipMemcpy(Output,d_update,ImSize*sizeof(float),hipMemcpyDeviceToHost);

            hipFree(d_input);
            hipFree(d_update);
            if (epsil != 0.0f) hipFree(d_update_prev);
            hipFree(P1);
            hipFree(P2);
            hipFree(P3);
            hipFree(P1_prev);
            hipFree(P2_prev);
            hipFree(P3_prev);
            hipFree(R1);
            hipFree(R2);
            hipFree(R3);
            hipFree(InputRef_x);
            hipFree(InputRef_y);
            hipFree(InputRef_z);
            hipFree(d_InputRef);
    }


    /*adding info into info_vector */
    infovector[0] = (float)(i);  /*iterations number (if stopped earlier based on tolerance)*/
    infovector[1] = re;  /* reached tolerance */
    hipDeviceSynchronize();
    return 0;
}
