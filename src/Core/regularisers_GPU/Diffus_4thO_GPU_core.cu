#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2017 Daniil Kazantsev
Copyright 2017 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "Diffus_4thO_GPU_core.h"
#include "shared.h"
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

/* CUDA implementation of fourth-order diffusion scheme [1] for piecewise-smooth recovery (2D/3D case)
 * The minimisation is performed using explicit scheme.
 *
 * Input Parameters:
 * 1. Noisy image/volume
 * 2. lambda - regularization parameter
 * 3. Edge-preserving parameter (sigma)
 * 4. Number of iterations, for explicit scheme >= 150 is recommended
 * 5. tau - time-marching step for explicit scheme
 * 6. eplsilon: tolerance constant
 * 7. GPU device number if for multigpu run (default 0)

 * Output:
 * [1] Filtered/regularized image/volume
 * [2] Information vector which contains [iteration no., reached tolerance]
 *
 * This function is based on the paper by
 * [1] Hajiaboli, M.R., 2011. An anisotropic fourth-order diffusion filter for image noise removal. International Journal of Computer Vision, 92(2), pp.177-191.
 */

#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8

#define BLKXSIZE2D 16
#define BLKYSIZE2D 16
#define EPS 1.0e-7
#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )
/********************************************************************/
/***************************2D Functions*****************************/
/********************************************************************/
__global__ void Weighted_Laplc2D_kernel(float *W_Lapl, float *U0, float sigma, int dimX, int dimY)
{
		int i1,i2,j1,j2;
		float gradX, gradX_sq, gradY, gradY_sq, gradXX, gradYY, gradXY, xy_2, denom, V_norm, V_orth, c, c_sq;

				int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        int index = i + dimX*j;

        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY)) {

            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == dimX) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == dimY) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;

				gradX = 0.5f*(U0[j*dimX+i2] - U0[j*dimX+i1]);
				gradX_sq = powf(gradX,2);

				gradY = 0.5f*(U0[j2*dimX+i] - U0[j1*dimX+i]);
                gradY_sq = powf(gradY,2);

                gradXX = U0[j*dimX+i2] + U0[j*dimX+i1] - 2*U0[index];
                gradYY = U0[j2*dimX+i] + U0[j1*dimX+i] - 2*U0[index];

                gradXY = 0.25f*(U0[j2*dimX+i2] + U0[j1*dimX+i1] - U0[j1*dimX+i2] - U0[j2*dimX+i1]);
                xy_2 = 2.0f*gradX*gradY*gradXY;

                denom =  gradX_sq + gradY_sq;

                if (denom <= EPS) {
                    V_norm = (gradXX*gradX_sq + xy_2 + gradYY*gradY_sq)/EPS;
                    V_orth = (gradXX*gradY_sq - xy_2 + gradYY*gradX_sq)/EPS;
                    }
                else  {
                    V_norm = (gradXX*gradX_sq + xy_2 + gradYY*gradY_sq)/denom;
                    V_orth = (gradXX*gradY_sq - xy_2 + gradYY*gradX_sq)/denom;
                    }

                c = 1.0f/(1.0f + denom/sigma);
                c_sq = c*c;

                W_Lapl[index] = c_sq*V_norm + c*V_orth;
		}
	return;
}

__global__ void Diffusion_update_step2D_kernel(float *Output, float *Input, float *W_Lapl, float lambdaPar, float sigmaPar2, float tau, int dimX, int dimY)
{
	int i1,i2,j1,j2;
    float gradXXc, gradYYc;

		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        int index = i + dimX*j;

        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY)) {

            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == dimX) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == dimY) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;

                    gradXXc = W_Lapl[j*dimX+i2] + W_Lapl[j*dimX+i1] - 2*W_Lapl[index];
                    gradYYc = W_Lapl[j2*dimX+i] + W_Lapl[j1*dimX+i] - 2*W_Lapl[index];

                    Output[index] += tau*(-lambdaPar*(gradXXc + gradYYc) - (Output[index] - Input[index]));
		}
	return;
}
/********************************************************************/
/***************************3D Functions*****************************/
/********************************************************************/
__global__ void Weighted_Laplc3D_kernel(float *W_Lapl, float *U0, float sigma, int dimX, int dimY, int dimZ)
{
		int i1,i2,j1,j2,k1,k2;
		float gradX, gradX_sq, gradY, gradY_sq, gradXX, gradYY, gradXY, xy_2, denom, V_norm, V_orth, c, c_sq, gradZ, gradZ_sq, gradZZ, gradXZ, gradYZ, xyz_1, xyz_2;

		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int k = blockDim.z * blockIdx.z + threadIdx.z;

		if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {

		    /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == dimX) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == dimY) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;
			k1 = k+1; if (k1 == dimZ) k1 = k-1;
			k2 = k-1; if (k2 < 0) k2 = k+1;

				int index = (dimX*dimY)*k + j*dimX+i;

				gradX = 0.5f*(U0[(dimX*dimY)*k + j*dimX+i2] - U0[(dimX*dimY)*k + j*dimX+i1]);
				gradX_sq = pow(gradX,2);

				gradY = 0.5f*(U0[(dimX*dimY)*k + j2*dimX+i] - U0[(dimX*dimY)*k + j1*dimX+i]);
                gradY_sq = pow(gradY,2);

                gradZ = 0.5f*(U0[(dimX*dimY)*k2 + j*dimX+i] - U0[(dimX*dimY)*k1 + j*dimX+i]);
                gradZ_sq = pow(gradZ,2);

                gradXX = U0[(dimX*dimY)*k + j*dimX+i2] + U0[(dimX*dimY)*k + j*dimX+i1] - 2*U0[index];
                gradYY = U0[(dimX*dimY)*k + j2*dimX+i] + U0[(dimX*dimY)*k + j1*dimX+i] - 2*U0[index];
                gradZZ = U0[(dimX*dimY)*k2 + j*dimX+i] + U0[(dimX*dimY)*k1 + j*dimX+i] - 2*U0[index];

                gradXY = 0.25f*(U0[(dimX*dimY)*k + j2*dimX+i2] + U0[(dimX*dimY)*k + j1*dimX+i1] - U0[(dimX*dimY)*k + j1*dimX+i2] - U0[(dimX*dimY)*k + j2*dimX+i1]);
                gradXZ = 0.25f*(U0[(dimX*dimY)*k2 + j*dimX+i2] - U0[(dimX*dimY)*k2+j*dimX+i1] - U0[(dimX*dimY)*k1+j*dimX+i2] + U0[(dimX*dimY)*k1+j*dimX+i1]);
                gradYZ = 0.25f*(U0[(dimX*dimY)*k2 +j2*dimX+i] - U0[(dimX*dimY)*k2+j1*dimX+i] - U0[(dimX*dimY)*k1+j2*dimX+i] + U0[(dimX*dimY)*k1+j1*dimX+i]);

                xy_2  = 2.0f*gradX*gradY*gradXY;
                xyz_1 = 2.0f*gradX*gradZ*gradXZ;
                xyz_2 = 2.0f*gradY*gradZ*gradYZ;

                denom =  gradX_sq + gradY_sq + gradZ_sq;

					if (denom <= EPS) {
					V_norm = (gradXX*gradX_sq + gradYY*gradY_sq + gradZZ*gradZ_sq + xy_2 + xyz_1 + xyz_2)/EPS;
                    V_orth = ((gradY_sq + gradZ_sq)*gradXX + (gradX_sq + gradZ_sq)*gradYY + (gradX_sq + gradY_sq)*gradZZ - xy_2 - xyz_1 - xyz_2)/EPS;
					}
					else  {
					V_norm = (gradXX*gradX_sq + gradYY*gradY_sq + gradZZ*gradZ_sq + xy_2 + xyz_1 + xyz_2)/denom;
                    V_orth = ((gradY_sq + gradZ_sq)*gradXX + (gradX_sq + gradZ_sq)*gradYY + (gradX_sq + gradY_sq)*gradZZ - xy_2 - xyz_1 - xyz_2)/denom;
					}

                c = 1.0f/(1.0f + denom/sigma);
                c_sq = c*c;

            W_Lapl[index] = c_sq*V_norm + c*V_orth;
		}
	return;
}
__global__ void Diffusion_update_step3D_kernel(float *Output, float *Input, float *W_Lapl, float lambdaPar, float sigmaPar2, float tau, int dimX, int dimY, int dimZ)
{
	int i1,i2,j1,j2,k1,k2;
    float gradXXc, gradYYc, gradZZc;

		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int k = blockDim.z * blockIdx.z + threadIdx.z;

		if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {

		    /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == dimX) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == dimY) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;
			k1 = k+1; if (k1 == dimZ) k1 = k-1;
			k2 = k-1; if (k2 < 0) k2 = k+1;

			int index = (dimX*dimY)*k + j*dimX+i;

                    gradXXc = W_Lapl[(dimX*dimY)*k + j*dimX+i2] + W_Lapl[(dimX*dimY)*k + j*dimX+i1] - 2*W_Lapl[index];
                    gradYYc = W_Lapl[(dimX*dimY)*k + j2*dimX+i] + W_Lapl[(dimX*dimY)*k + j1*dimX+i] - 2*W_Lapl[index];
                    gradZZc = W_Lapl[(dimX*dimY)*k2 + j*dimX+i] + W_Lapl[(dimX*dimY)*k1 + j*dimX+i] - 2*W_Lapl[index];

                    Output[index] += tau*(-lambdaPar*(gradXXc + gradYYc + gradZZc) - (Output[index] - Input[index]));
		}
	return;
}

__global__ void Diff4thcopy_kernel2D(float *Input, float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        Output[index] = Input[index];
    }
}


__global__ void Diff4thResidCalc2D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        Output[index] = Input1[index] - Input2[index];
    }
}

__global__ void Diff4thcopy_kernel3D(float *Input, float* Output, int N, int M, int Z, int num_total)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        Output[index] = Input[index];
    }
}

__global__ void Diff4thResidCalc3D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int Z, int num_total)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        Output[index] = Input1[index] - Input2[index];
    }
}

/*%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%*/
/********************* MAIN HOST FUNCTION ******************/
/*%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%*/
extern "C" int Diffus4th_GPU_main(float *Input, float *Output, float *infovector, float lambdaPar, float sigmaPar, int iterationsNumb, float tau, float epsil, int gpu_device, int N, int M, int Z)
{

    int deviceCount = -1; // number of devices
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        return -1;
      }
        
        checkCudaErrors(hipSetDevice(gpu_device)); 

        int dimTotal, n, count = 0;
        float *d_input, *d_output, *d_W_Lapl, *d_update_prev=NULL, re;
        re = 0.0f;
        float sigmaPar2;
        sigmaPar2 = sigmaPar*sigmaPar;
        dimTotal = N*M*Z;

        CHECK(hipMalloc((void**)&d_input,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&d_output,dimTotal*sizeof(float)));
        CHECK(hipMalloc((void**)&d_W_Lapl,dimTotal*sizeof(float)));
        if (epsil != 0.0f) checkCudaErrors( hipMalloc((void**)&d_update_prev,dimTotal*sizeof(float)) );

        CHECK(hipMemcpy(d_input,Input,dimTotal*sizeof(float),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_output,Input,dimTotal*sizeof(float),hipMemcpyHostToDevice));

       /*2D case */
       dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
       dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));
       dim3 dimBlock3(BLKXSIZE,BLKYSIZE,BLKZSIZE);
       dim3 dimGrid3(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKZSIZE));


        for(n=0; n < iterationsNumb; n++) {

					if ((epsil != 0.0f) && (n % 5 == 0)) {
						if (Z == 1) Diff4thcopy_kernel2D<<<dimGrid,dimBlock>>>(d_output, d_update_prev, N, M, dimTotal);
						else Diff4thcopy_kernel3D<<<dimGrid3,dimBlock3>>>(d_output, d_update_prev, N, M, Z, dimTotal);
						checkCudaErrors( hipDeviceSynchronize() );
						checkCudaErrors(hipPeekAtLastError() );
					}

          if (Z == 1) {
        	     /*2D case */
				       /* Calculating weighted Laplacian */
				      Weighted_Laplc2D_kernel<<<dimGrid,dimBlock>>>(d_W_Lapl, d_output, sigmaPar2, N, M);
				      CHECK(hipDeviceSynchronize());
				      /* Perform iteration step */
				      Diffusion_update_step2D_kernel<<<dimGrid,dimBlock>>>(d_output, d_input, d_W_Lapl, lambdaPar, sigmaPar2, tau, N, M);
				      CHECK(hipDeviceSynchronize());
                }
            else {
              /* Calculating weighted Laplacian */
              Weighted_Laplc3D_kernel<<<dimGrid3,dimBlock3>>>(d_W_Lapl, d_output, sigmaPar2, N, M, Z);
              CHECK(hipDeviceSynchronize());
              /* Perform iteration step */
              Diffusion_update_step3D_kernel<<<dimGrid3,dimBlock3>>>(d_output, d_input, d_W_Lapl, lambdaPar, sigmaPar2, tau, N, M, Z);
              CHECK(hipDeviceSynchronize());
            }
            if ((epsil != 0.0f) && (n % 5 == 0)) {
            /* calculate norm - stopping rules using the Thrust library */
            if (Z == 1) Diff4thResidCalc2D_kernel<<<dimGrid,dimBlock>>>(d_output, d_update_prev, d_W_Lapl, N, M, dimTotal);
            else Diff4thResidCalc3D_kernel<<<dimGrid3,dimBlock3>>>(d_output, d_update_prev, d_W_Lapl, N, M, Z, dimTotal);
            checkCudaErrors( hipDeviceSynchronize() );
            checkCudaErrors( hipPeekAtLastError() );

            // setup arguments
            square<float>        unary_op;
            thrust::plus<float> binary_op;
            thrust::device_vector<float> d_vec(d_W_Lapl, d_W_Lapl + dimTotal);
            float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
            thrust::device_vector<float> d_vec2(d_output, d_output + dimTotal);
            float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

            // compute norm
            re = (reduction/reduction2);
            if (re < epsil)  count++;
            if (count > 3) break;
            }
	       }

        CHECK(hipMemcpy(Output,d_output,dimTotal*sizeof(float),hipMemcpyDeviceToHost));
        CHECK(hipFree(d_input));
        CHECK(hipFree(d_output));
        CHECK(hipFree(d_W_Lapl));
        if (epsil != 0.0f) hipFree(d_update_prev);

        /*adding info into info_vector */
        infovector[0] = (float)(n);  /*iterations number (if stopped earlier based on tolerance)*/
        infovector[1] = re;  /* reached tolerance */
        hipDeviceSynchronize();
        return 0;
}
