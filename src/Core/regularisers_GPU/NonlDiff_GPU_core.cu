#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2017 Daniil Kazantsev
Copyright 2017 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "NonlDiff_GPU_core.h"
#include "shared.h"
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

/* CUDA implementation of linear and nonlinear diffusion with the regularisation model [1,2] (2D/3D case)
 * The minimisation is performed using explicit scheme.
 *
 * Input Parameters:
 * 1. Noisy image/volume
 * 2. lambda - regularization parameter
 * 3. Edge-preserving parameter (sigma), when sigma equals to zero nonlinear diffusion -> linear diffusion
 * 4. Number of iterations, for explicit scheme >= 150 is recommended
 * 5. tau - time-marching step for explicit scheme
 * 6. Penalty type: 1 - Huber, 2 - Perona-Malik, 3 - Tukey Biweight, 4 - Threshold-constrained Linear, 5 - modified Huber with a dead stop on edge
 * 7. eplsilon: tolerance constant
 * 8. GPU device number if for multigpu run (default 0)

  * Output:
  * [1] Filtered/regularized image/volume
  * [2] Information vector which contains [iteration no., reached tolerance]
 *
 * This function is based on the paper by
 * [1] Perona, P. and Malik, J., 1990. Scale-space and edge detection using anisotropic diffusion. IEEE Transactions on pattern analysis and machine intelligence, 12(7), pp.629-639.
 * [2] Black, M.J., Sapiro, G., Marimont, D.H. and Heeger, D., 1998. Robust anisotropic diffusion. IEEE Transactions on image processing, 7(3), pp.421-432.
 */

#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8

#define BLKXSIZE2D 16
#define BLKYSIZE2D 16
#define EPS 1.0e-5

#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__host__ __device__ int signNDF (float x)
{
        return (x > 0) - (x < 0);
}

/********************************************************************/
/***************************2D Functions*****************************/
/********************************************************************/
__global__ void LinearDiff2D_kernel(float *Input, float *Output, float lambdaPar, float tau, int N, int M)
    {
		int i1,i2,j1,j2;
		float e,w,n,s,e1,w1,n1,s1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        int index = i + N*j;

        if ((i >= 0) && (i < N) && (j >= 0) && (j < M)) {

            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == N) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == M) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;

		        e = Output[j*N+i1];
                w = Output[j*N+i2];
                n = Output[j1*N+i];
                s = Output[j2*N+i];

                e1 = e - Output[index];
                w1 = w - Output[index];
                n1 = n - Output[index];
                s1 = s - Output[index];

                Output[index] += tau*(lambdaPar*(e1 + w1 + n1 + s1) - (Output[index] - Input[index]));
		}
	}

 __global__ void NonLinearDiff2D_kernel(float *Input, float *Output, float lambdaPar, float sigmaPar, float tau, int penaltytype, int N, int M)
    {
		int i1,i2,j1,j2;
		float e,w,n,s,e1,w1,n1,s1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        int index = i + N*j;

        if ((i >= 0) && (i < N) && (j >= 0) && (j < M)) {

            /* boundary conditions (Neumann reflections) */
			      i1 = i+1; if (i1 == N) i1 = i-1;
			      i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == M) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;

		        e = Output[j*N+i1];
                w = Output[j*N+i2];
                n = Output[j1*N+i];
                s = Output[j2*N+i];

                e1 = e - Output[index];
                w1 = w - Output[index];
                n1 = n - Output[index];
                s1 = s - Output[index];

            if (penaltytype == 1){
            /* Huber penalty */
            if (abs(e1) > sigmaPar) e1 =  signNDF(e1);
            else e1 = e1/sigmaPar;

            if (abs(w1) > sigmaPar) w1 =  signNDF(w1);
            else w1 = w1/sigmaPar;

            if (abs(n1) > sigmaPar) n1 =  signNDF(n1);
            else n1 = n1/sigmaPar;

            if (abs(s1) > sigmaPar) s1 =  signNDF(s1);
            else s1 = s1/sigmaPar;
            }
            else if (penaltytype == 2) {
            /* Perona-Malik */
            e1 = (e1)/(1.0f + pow((e1/sigmaPar),2));
            w1 = (w1)/(1.0f + pow((w1/sigmaPar),2));
            n1 = (n1)/(1.0f + pow((n1/sigmaPar),2));
            s1 = (s1)/(1.0f + pow((s1/sigmaPar),2));
            }
            else if (penaltytype == 3) {
            /* Tukey Biweight */
            if (abs(e1) <= sigmaPar) e1 =  e1*pow((1.0f - pow((e1/sigmaPar),2)), 2);
            else e1 = 0.0f;
            if (abs(w1) <= sigmaPar) w1 =  w1*pow((1.0f - pow((w1/sigmaPar),2)), 2);
            else w1 = 0.0f;
            if (abs(n1) <= sigmaPar) n1 =  n1*pow((1.0f - pow((n1/sigmaPar),2)), 2);
            else n1 = 0.0f;
            if (abs(s1) <= sigmaPar) s1 =  s1*pow((1.0f - pow((s1/sigmaPar),2)), 2);
            else s1 = 0.0f;
            }
            else if (penaltytype == 4) {
                /* Threshold-constrained linear diffusion
                This means that the linear diffusion will be performed on pixels with
                absolute difference less than the threshold.
                */
                if (abs(e1) > sigmaPar) e1 = 0.0f;
                if (abs(w1) > sigmaPar) w1 = 0.0f;
                if (abs(n1) > sigmaPar) n1 = 0.0f;
                if (abs(s1) > sigmaPar) s1 = 0.0f;
            }
            else if (penaltytype == 5) {
                /* Threshold-constrained Huber nonlinear diffusion
                This means that the linear diffusion will be performed on pixels with
                absolute difference less than the threshold.
                */
                if (abs(e1) <= 2.0f*sigmaPar) {
                if (abs(e1) > sigmaPar) e1 =  signNDF(e1);
                else e1 = e1/sigmaPar;}
                else e1 = 0.0f;

                if (abs(w1) <= 2.0f*sigmaPar) {
                if (abs(w1) > sigmaPar) w1 =  signNDF(w1);
                else w1 = w1/sigmaPar;}
                else w1 = 0.0f;

                if (abs(n1) <= 2.0f*sigmaPar) {
                if (abs(n1) > sigmaPar) n1 =  signNDF(n1);
                else n1 = n1/sigmaPar; }
                else n1 = 0.0f;

                if (abs(s1) <= 2.0f*sigmaPar) {
                if (abs(s1) > sigmaPar) s1 =  signNDF(s1);
                else s1 = s1/sigmaPar; }
                else s1 = 0.0f;
            }
            else printf("%s \n", "No penalty function selected! Use 1,2,3, 4 or 5.");

            Output[index] += tau*(lambdaPar*(e1 + w1 + n1 + s1) - (Output[index] - Input[index]));
		}
	}
/********************************************************************/
/***************************3D Functions*****************************/
/********************************************************************/

__global__ void LinearDiff3D_kernel(float *Input, float *Output, float lambdaPar, float tau, int N, int M, int Z)
    {
		int i1,i2,j1,j2,k1,k2;
		float e,w,n,s,u,d,e1,w1,n1,s1,u1,d1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int k = blockDim.z * blockIdx.z + threadIdx.z;

		int index = (N*M)*k + i + N*j;

        if ((i >= 0) && (i < N) && (j >= 0) && (j < M) && (k >= 0) && (k < Z)) {

            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == N) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == M) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;
			k1 = k+1; if (k1 == Z) k1 = k-1;
			k2 = k-1; if (k2 < 0) k2 = k+1;

		        e = Output[(N*M)*k + i1 + N*j];
                w = Output[(N*M)*k + i2 + N*j];
                n = Output[(N*M)*k + i + N*j1];
                s = Output[(N*M)*k + i + N*j2];
                u = Output[(N*M)*k1 + i + N*j];
                d = Output[(N*M)*k2 + i + N*j];

                e1 = e - Output[index];
                w1 = w - Output[index];
                n1 = n - Output[index];
                s1 = s - Output[index];
                u1 = u - Output[index];
                d1 = d - Output[index];

                Output[index] += tau*(lambdaPar*(e1 + w1 + n1 + s1 + u1 + d1) - (Output[index] - Input[index]));
		}
	}

__global__ void NonLinearDiff3D_kernel(float *Input, float *Output, float lambdaPar, float sigmaPar, float tau, int penaltytype, int N, int M, int Z)
    {
		int i1,i2,j1,j2,k1,k2;
		float e,w,n,s,u,d,e1,w1,n1,s1,u1,d1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		int j = blockDim.y * blockIdx.y + threadIdx.y;
		int k = blockDim.z * blockIdx.z + threadIdx.z;

		int index = (N*M)*k + i + N*j;

        if ((i >= 0) && (i < N) && (j >= 0) && (j < M) && (k >= 0) && (k < Z)) {

            /* boundary conditions (Neumann reflections) */
			i1 = i+1; if (i1 == N) i1 = i-1;
			i2 = i-1; if (i2 < 0) i2 = i+1;
            j1 = j+1; if (j1 == M) j1 = j-1;
            j2 = j-1; if (j2 < 0) j2 = j+1;
			k1 = k+1; if (k1 == Z) k1 = k-1;
			k2 = k-1; if (k2 < 0) k2 = k+1;

		        e = Output[(N*M)*k + i1 + N*j];
                w = Output[(N*M)*k + i2 + N*j];
                n = Output[(N*M)*k + i + N*j1];
                s = Output[(N*M)*k + i + N*j2];
                u = Output[(N*M)*k1 + i + N*j];
                d = Output[(N*M)*k2 + i + N*j];

                e1 = e - Output[index];
                w1 = w - Output[index];
                n1 = n - Output[index];
                s1 = s - Output[index];
                u1 = u - Output[index];
                d1 = d - Output[index];


            if (penaltytype == 1){
            /* Huber penalty */
            if (abs(e1) > sigmaPar) e1 =  signNDF(e1);
            else e1 = e1/sigmaPar;

            if (abs(w1) > sigmaPar) w1 =  signNDF(w1);
            else w1 = w1/sigmaPar;

            if (abs(n1) > sigmaPar) n1 =  signNDF(n1);
            else n1 = n1/sigmaPar;

            if (abs(s1) > sigmaPar) s1 =  signNDF(s1);
            else s1 = s1/sigmaPar;

            if (abs(u1) > sigmaPar) u1 =  signNDF(u1);
            else u1 = u1/sigmaPar;

            if (abs(d1) > sigmaPar) d1 =  signNDF(d1);
            else d1 = d1/sigmaPar;
            }
            else if (penaltytype == 2) {
            /* Perona-Malik */
            e1 = (e1)/(1.0f + pow((e1/sigmaPar),2));
            w1 = (w1)/(1.0f + pow((w1/sigmaPar),2));
            n1 = (n1)/(1.0f + pow((n1/sigmaPar),2));
            s1 = (s1)/(1.0f + pow((s1/sigmaPar),2));
            u1 = (u1)/(1.0f + pow((u1/sigmaPar),2));
            d1 = (d1)/(1.0f + pow((d1/sigmaPar),2));
            }
            else if (penaltytype == 3) {
            /* Tukey Biweight */
            if (abs(e1) <= sigmaPar) e1 =  e1*pow((1.0f - pow((e1/sigmaPar),2)), 2);
            else e1 = 0.0f;
            if (abs(w1) <= sigmaPar) w1 =  w1*pow((1.0f - pow((w1/sigmaPar),2)), 2);
            else w1 = 0.0f;
            if (abs(n1) <= sigmaPar) n1 =  n1*pow((1.0f - pow((n1/sigmaPar),2)), 2);
            else n1 = 0.0f;
            if (abs(s1) <= sigmaPar) s1 =  s1*pow((1.0f - pow((s1/sigmaPar),2)), 2);
            else s1 = 0.0f;
            if (abs(u1) <= sigmaPar) u1 =  u1*pow((1.0f - pow((u1/sigmaPar),2)), 2);
            else u1 = 0.0f;
            if (abs(d1) <= sigmaPar) d1 =  d1*pow((1.0f - pow((d1/sigmaPar),2)), 2);
            else d1 = 0.0f;
            }
            else if (penaltytype == 4) {
                /* Threshold-constrained linear diffusion
                This means that the linear diffusion will be performed on pixels with
                absolute difference less than the threshold.
                */
                if (abs(e1) > sigmaPar) e1 = 0.0f;
                if (abs(w1) > sigmaPar) w1 = 0.0f;
                if (abs(n1) > sigmaPar) n1 = 0.0f;
                if (abs(s1) > sigmaPar) s1 = 0.0f;
                if (abs(u1) > sigmaPar) u1 = 0.0f;
                if (abs(d1) > sigmaPar) d1 = 0.0f;
            }
            else if (penaltytype == 5) {
                /* Threshold-constrained Huber nonlinear diffusion
                This means that the linear diffusion will be performed on pixels with
                absolute difference less than the threshold.
                */
                if (abs(e1) <= 2.0f*sigmaPar) {
                if (abs(e1) > sigmaPar) e1 =  signNDF(e1);
                else e1 = e1/sigmaPar;}
                else e1 = 0.0f;

                if (abs(w1) <= 2.0f*sigmaPar) {
                if (abs(w1) > sigmaPar) w1 =  signNDF(w1);
                else w1 = w1/sigmaPar;}
                else w1 = 0.0f;

                if (abs(n1) <= 2.0f*sigmaPar) {
                if (abs(n1) > sigmaPar) n1 =  signNDF(n1);
                else n1 = n1/sigmaPar; }
                else n1 = 0.0f;

                if (abs(s1) <= 2.0f*sigmaPar) {
                if (abs(s1) > sigmaPar) s1 =  signNDF(s1);
                else s1 = s1/sigmaPar; }
                else s1 = 0.0f;

                if (abs(u1) <= 2.0f*sigmaPar) {
                if (abs(u1) > sigmaPar) u1 =  signNDF(u1);
                else u1 = u1/sigmaPar; }
                else u1 = 0.0f;

                if (abs(d1) <= 2.0f*sigmaPar) {
                if (abs(d1) > sigmaPar) d1 =  signNDF(d1);
                else d1 = d1/sigmaPar; }
                else d1 = 0.0f;
            }
            else printf("%s \n", "No penalty function selected! Use 1,2,3,4, or 5.");
            Output[index] += tau*(lambdaPar*(e1 + w1 + n1 + s1 + u1 + d1) - (Output[index] - Input[index]));
		}
	}

  __global__ void NDFcopy_kernel2D(float *Input, float* Output, int N, int M, int num_total)
  {
      int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
      int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

      int index = xIndex + N*yIndex;

      if (index < num_total)	{
          Output[index] = Input[index];
      }
  }
  __global__ void NDFResidCalc2D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int num_total)
  {
      int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
      int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

      int index = xIndex + N*yIndex;

      if (index < num_total)	{
          Output[index] = Input1[index] - Input2[index];
      }
  }
  __global__ void NDFcopy_kernel3D(float *Input, float* Output, int N, int M, int Z, int num_total)
  {
  	int i = blockDim.x * blockIdx.x + threadIdx.x;
      int j = blockDim.y * blockIdx.y + threadIdx.y;
      int k = blockDim.z * blockIdx.z + threadIdx.z;

      int index = (N*M)*k + i + N*j;

      if (index < num_total)	{
          Output[index] = Input[index];
      }
  }
  __global__ void NDFResidCalc3D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int Z, int num_total)
  {
  	int i = blockDim.x * blockIdx.x + threadIdx.x;
      int j = blockDim.y * blockIdx.y + threadIdx.y;
      int k = blockDim.z * blockIdx.z + threadIdx.z;

      int index = (N*M)*k + i + N*j;

      if (index < num_total)	{
          Output[index] = Input1[index] - Input2[index];
      }
  }


/////////////////////////////////////////////////
// HOST FUNCTION
extern "C" int NonlDiff_GPU_main(float *Input, float *Output, float *infovector, float lambdaPar, float sigmaPar, int iterationsNumb, float tau, int penaltytype, float epsil, int gpu_device, int N, int M, int Z)
{
  int deviceCount = -1; // number of devices
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "No CUDA devices found\n");
       return -1;
   }
        checkCudaErrors(hipSetDevice(gpu_device));

        int n, count, ImSize;
        count = 0;
        float *d_input, *d_output, *d_update_prev, *d_res;
        float sigmaPar2, re = 0.0f;
        sigmaPar2 = sigmaPar/sqrt(2.0f);
	      ImSize = N*M*Z;


        CHECK(hipMalloc((void**)&d_input,ImSize*sizeof(float)));
        CHECK(hipMalloc((void**)&d_output,ImSize*sizeof(float)));
        if (epsil != 0.0f) {
        checkCudaErrors( hipMalloc((void**)&d_update_prev,ImSize*sizeof(float)) );
        checkCudaErrors( hipMalloc((void**)&d_res,ImSize*sizeof(float)) );
        }

        CHECK(hipMemcpy(d_input,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_output,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));

	if (Z == 1) {
	     /*2D case */

        dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));

        for(n=0; n < iterationsNumb; n++) {

        if ((epsil != 0.0f) && (n % 5 == 0)) {
        NDFcopy_kernel2D<<<dimGrid,dimBlock>>>(d_output, d_update_prev, N, M, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
        }

				if (sigmaPar == 0.0f) {
				/* linear diffusion (heat equation) */
				LinearDiff2D_kernel<<<dimGrid,dimBlock>>>(d_input, d_output, lambdaPar, tau, N, M);
				CHECK(hipDeviceSynchronize());
				}
				else {
				/* nonlinear diffusion */
				NonLinearDiff2D_kernel<<<dimGrid,dimBlock>>>(d_input, d_output, lambdaPar, sigmaPar2, tau, penaltytype, N, M);
				CHECK(hipDeviceSynchronize());
				}

        if ((epsil != 0.0f) && (n % 5 == 0)) {
        /* calculate norm - stopping rules using the Thrust library */
        NDFResidCalc2D_kernel<<<dimGrid,dimBlock>>>(d_output, d_update_prev, d_res, N, M, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors( hipPeekAtLastError() );

        // setup arguments
        square<float>        unary_op;
        thrust::plus<float> binary_op;
        thrust::device_vector<float> d_vec(d_res, d_res + ImSize);
        float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
        thrust::device_vector<float> d_vec2(d_output, d_output + ImSize);
        float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

        // compute norm
        re = (reduction/reduction2);
        if (re < epsil)  count++;
        if (count > 3) break;
          }
        }
	}
	else {
		/*3D case*/
        dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
        dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKZSIZE));
			for(n=0; n < iterationsNumb; n++) {

        if ((epsil != 0.0f) && (n % 5 == 0)) {
        NDFcopy_kernel3D<<<dimGrid,dimBlock>>>(d_output, d_update_prev, N, M, Z, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
        }

				if (sigmaPar == 0.0f) {
				/* linear diffusion (heat equation) */
				LinearDiff3D_kernel<<<dimGrid,dimBlock>>>(d_input, d_output, lambdaPar, tau, N, M, Z);
				CHECK(hipDeviceSynchronize());
				}
				else {
				/* nonlinear diffusion */
				NonLinearDiff3D_kernel<<<dimGrid,dimBlock>>>(d_input, d_output, lambdaPar, sigmaPar2, tau, penaltytype, N, M, Z);
				CHECK(hipDeviceSynchronize());
				}

        if ((epsil != 0.0f) && (n % 5 == 0)) {
        /* calculate norm - stopping rules using the Thrust library */
        NDFResidCalc3D_kernel<<<dimGrid,dimBlock>>>(d_output, d_update_prev, d_res, N, M, Z, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors( hipPeekAtLastError() );

        // setup arguments
        square<float>        unary_op;
        thrust::plus<float> binary_op;
        thrust::device_vector<float> d_vec(d_res, d_res + ImSize);
        float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
        thrust::device_vector<float> d_vec2(d_output, d_output + ImSize);
        float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

        // compute norm
        re = (reduction/reduction2);
        if (re < epsil)  count++;
        if (count > 3) break;
          }
			}

		}
        CHECK(hipMemcpy(Output,d_output,ImSize*sizeof(float),hipMemcpyDeviceToHost));
        CHECK(hipFree(d_input));
        CHECK(hipFree(d_output));
        if (epsil != 0.0f) {
        CHECK(hipFree(d_update_prev));
        CHECK(hipFree(d_res));
        }

        infovector[0] = (float)(n);  /*iterations number (if stopped earlier based on tolerance)*/
        infovector[1] = re;  /* reached tolerance */
        hipDeviceSynchronize();
        return 0;
}
