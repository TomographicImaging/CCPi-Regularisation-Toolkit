#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2019 Daniil Kazantsev
Copyright 2019 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "TV_ROF_GPU_core.h"
#include "shared.h"
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

/* C-OMP implementation of ROF-TV denoising/regularization model [1] (2D/3D case)
*
* Input Parameters:
* 1. Noisy image/volume [REQUIRED]
* 2. lambda - regularization parameter (a constant or the same size as input (1))
* 3. tau - marching step for explicit scheme, ~1 is recommended [REQUIRED]
* 4. Number of iterations, for explicit scheme >= 150 is recommended  [REQUIRED]
* 5. eplsilon: tolerance constant
* 6. GPU device number if for multigpu run (default 0)

* Output:
* [1] Regularised image/volume
* [2] Information vector which contains [iteration no., reached tolerance]
*
* This function is based on the paper by
* [1] Rudin, Osher, Fatemi, "Nonlinear Total Variation based noise removal algorithms"
*/

#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8

#define BLKXSIZE2D 16
#define BLKYSIZE2D 16
#define EPS 1.0e-8

#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__host__ __device__ int sign (float x)
{
        return (x > 0) - (x < 0);
}

/*********************2D case****************************/

    /* differences 1 */
    __global__ void D1_func2D(float* Input, float* D1, int N, int M)
    {
		int i1, j1, i2;
		float NOMx_1,NOMy_1,NOMy_0,denom1,denom2,T1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        int index = i + N*j;

        if ((i >= 0) && (i < N) && (j >= 0) && (j < M)) {

            /* boundary conditions (Neumann reflections) */
                i1 = i + 1; if (i1 >= N) i1 = i-1;
                i2 = i - 1; if (i2 < 0) i2 = i+1;
                j1 = j + 1; if (j1 >= M) j1 = j-1;

		     /* Forward-backward differences */
                NOMx_1 = Input[j1*N + i] - Input[index]; /* x+ */
                NOMy_1 = Input[j*N + i1] - Input[index]; /* y+ */
                NOMy_0 = Input[index] - Input[j*N + i2]; /* y- */

                denom1 = NOMx_1*NOMx_1;
                denom2 = 0.5f*(sign((float)NOMy_1) + sign((float)NOMy_0))*(MIN(abs((float)NOMy_1), abs((float)NOMy_0)));
                denom2 = denom2*denom2;
                T1 = sqrt(denom1 + denom2 + EPS);
                D1[index] = NOMx_1/T1;
		}
	}

    /* differences 2 */
    __global__ void D2_func2D(float* Input, float* D2, int N, int M)
    {
		int i1, j1, j2;
		float NOMx_1,NOMy_1,NOMx_0,denom1,denom2,T2;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;

        int index = i + N*j;

        if ((i >= 0) && (i < (N)) && (j >= 0) && (j < (M))) {

            /* boundary conditions (Neumann reflections) */
                i1 = i + 1; if (i1 >= N) i1 = i-1;
                j1 = j + 1; if (j1 >= M) j1 = j-1;
                j2 = j - 1; if (j2 < 0) j2 = j+1;

                /* Forward-backward differences */
                NOMx_1 = Input[j1*N + i] - Input[index]; /* x+ */
                NOMy_1 = Input[j*N + i1] - Input[index]; /* y+ */
                NOMx_0 = Input[index] - Input[j2*N + i]; /* x- */

                denom1 = NOMy_1*NOMy_1;
                denom2 = 0.5f*(sign((float)NOMx_1) + sign((float)NOMx_0))*(MIN(abs((float)NOMx_1), abs((float)NOMx_0)));
                denom2 = denom2*denom2;
                T2 = sqrt(denom1 + denom2 + EPS);
                D2[index] = NOMy_1/T2;
		}
	}

    __global__ void TV_kernel2D(float *D1, float *D2, float *Update, float *Input, float *lambdaPar_d, int lambda_is_arr, float tau, int N, int M)
    {
		int i2, j2;
		float dv1,dv2,lambda_val;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int index = i + N*j;
    lambda_val = *(lambdaPar_d + index* lambda_is_arr);

        if ((i >= 0) && (i < (N)) && (j >= 0) && (j < (M))) {
				/* boundary conditions (Neumann reflections) */
                i2 = i - 1; if (i2 < 0) i2 = i+1;
                j2 = j - 1; if (j2 < 0) j2 = j+1;
				/* divergence components  */
                dv1 = D1[index] - D1[j2*N + i];
                dv2 = D2[index] - D2[j*N + i2];

                Update[index] += tau*(lambda_val*(dv1 + dv2) - (Update[index] - Input[index]));
		}
	}
/*********************3D case****************************/

    /* differences 1 */
    __global__ void D1_func3D(float* Input, float* D1, int dimX, int dimY, int dimZ)
    {
		float NOMx_1, NOMy_1, NOMy_0, NOMz_1, NOMz_0, denom1, denom2,denom3, T1;
		int i1,i2,k1,j1,j2,k2;

		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;

      	int index = (dimX*dimY)*k + j*dimX+i;

        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {

                    /* symmetric boundary conditions (Neuman) */
                    i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                    i2 = i - 1; if (i2 < 0) i2 = i+1;
                    j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                    j2 = j - 1; if (j2 < 0) j2 = j+1;
                    k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                    k2 = k - 1; if (k2 < 0) k2 = k+1;

                    /* Forward-backward differences */
                    NOMx_1 = Input[(dimX*dimY)*k + j1*dimX + i] - Input[index]; /* x+ */
                    NOMy_1 = Input[(dimX*dimY)*k + j*dimX + i1] - Input[index]; /* y+ */
                    NOMy_0 = Input[index] - Input[(dimX*dimY)*k + j*dimX + i2]; /* y- */

                    NOMz_1 = Input[(dimX*dimY)*k1 + j*dimX + i] - Input[index]; /* z+ */
                    NOMz_0 = Input[index] - Input[(dimX*dimY)*k2 + j*dimX + i]; /* z- */


                    denom1 = NOMx_1*NOMx_1;
                    denom2 = 0.5*(sign(NOMy_1) + sign(NOMy_0))*(MIN(abs(NOMy_1),abs(NOMy_0)));
                    denom2 = denom2*denom2;
                    denom3 = 0.5*(sign(NOMz_1) + sign(NOMz_0))*(MIN(abs(NOMz_1),abs(NOMz_0)));
                    denom3 = denom3*denom3;
                    T1 = sqrt(denom1 + denom2 + denom3 + EPS);
                    D1[index] = NOMx_1/T1;
		}
	}

    /* differences 2 */
    __global__ void D2_func3D(float* Input, float* D2, int dimX, int dimY, int dimZ)
    {
		float NOMx_1, NOMy_1, NOMx_0, NOMz_1, NOMz_0, denom1, denom2, denom3, T2;
		int i1,i2,k1,j1,j2,k2;

		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;

      	int index = (dimX*dimY)*k + j*dimX+i;

        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {
                    /* symmetric boundary conditions (Neuman) */
                    i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                    i2 = i - 1; if (i2 < 0) i2 = i+1;
                    j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                    j2 = j - 1; if (j2 < 0) j2 = j+1;
                    k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                    k2 = k - 1; if (k2 < 0) k2 = k+1;


                    /* Forward-backward differences */
                    NOMx_1 = Input[(dimX*dimY)*k + (j1)*dimX + i] - Input[index]; /* x+ */
                    NOMy_1 = Input[(dimX*dimY)*k + (j)*dimX + i1] - Input[index]; /* y+ */
                    NOMx_0 = Input[index] - Input[(dimX*dimY)*k + (j2)*dimX + i]; /* x- */
                    NOMz_1 = Input[(dimX*dimY)*k1 + j*dimX + i] - Input[index]; /* z+ */
                    NOMz_0 = Input[index] - Input[(dimX*dimY)*k2 + (j)*dimX + i]; /* z- */


                    denom1 = NOMy_1*NOMy_1;
                    denom2 = 0.5*(sign(NOMx_1) + sign(NOMx_0))*(MIN(abs(NOMx_1),abs(NOMx_0)));
                    denom2 = denom2*denom2;
                    denom3 = 0.5*(sign(NOMz_1) + sign(NOMz_0))*(MIN(abs(NOMz_1),abs(NOMz_0)));
                    denom3 = denom3*denom3;
                    T2 = sqrt(denom1 + denom2 + denom3 + EPS);
                    D2[index] = NOMy_1/T2;
		}
	}

	  /* differences 3 */
    __global__ void D3_func3D(float* Input, float* D3, int dimX, int dimY, int dimZ)
    {
		float NOMx_1, NOMy_1, NOMx_0, NOMy_0, NOMz_1, denom1, denom2, denom3, T3;
		int i1,i2,k1,j1,j2,k2;

		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;

      	int index = (dimX*dimY)*k + j*dimX+i;

        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {

				i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                i2 = i - 1; if (i2 < 0) i2 = i+1;
                j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                j2 = j - 1; if (j2 < 0) j2 = j+1;
                k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                k2 = k - 1; if (k2 < 0) k2 = k+1;

                /* Forward-backward differences */
                NOMx_1 = Input[(dimX*dimY)*k + (j1)*dimX + i] - Input[index]; /* x+ */
                NOMy_1 = Input[(dimX*dimY)*k + (j)*dimX + i1] - Input[index]; /* y+ */
                NOMy_0 = Input[index] - Input[(dimX*dimY)*k + (j)*dimX + i2]; /* y- */
                NOMx_0 = Input[index] - Input[(dimX*dimY)*k + (j2)*dimX + i]; /* x- */
                NOMz_1 = Input[(dimX*dimY)*k1 + j*dimX + i] - Input[index]; /* z+ */

                denom1 = NOMz_1*NOMz_1;
                denom2 = 0.5*(sign(NOMx_1) + sign(NOMx_0))*(MIN(abs(NOMx_1),abs(NOMx_0)));
                denom2 = denom2*denom2;
                denom3 = 0.5*(sign(NOMy_1) + sign(NOMy_0))*(MIN(abs(NOMy_1),abs(NOMy_0)));
                denom3 = denom3*denom3;
                T3 = sqrt(denom1 + denom2 + denom3 + EPS);
                D3[index] = NOMz_1/T3;
		}
	}

    __global__ void TV_kernel3D(float *D1, float *D2, float *D3, float *Update, float *Input, float *lambda, int lambda_is_arr, float tau, int dimX, int dimY, int dimZ)
    {
		float dv1, dv2, dv3, lambda_val;
		int i1,i2,k1,j1,j2,k2;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (dimX*dimY)*k + j*dimX+i;
    lambda_val = *(lambda + index* lambda_is_arr);

    if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {

		/* symmetric boundary conditions (Neuman) */
                    i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                    i2 = i - 1; if (i2 < 0) i2 = i+1;
                    j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                    j2 = j - 1; if (j2 < 0) j2 = j+1;
                    k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                    k2 = k - 1; if (k2 < 0) k2 = k+1;

                    /*divergence components */
                    dv1 = D1[index] - D1[(dimX*dimY)*k + j2*dimX+i];
                    dv2 = D2[index] - D2[(dimX*dimY)*k + j*dimX+i2];
                    dv3 = D3[index] - D3[(dimX*dimY)*k2 + j*dimX+i];

                    Update[index] += tau*(lambda_val*(dv1 + dv2 + dv3) - (Update[index] - Input[index]));

		}
	}
__global__ void ROFcopy_kernel2D(float *Input, float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        Output[index] = Input[index];
    }
}


__global__ void ROFResidCalc2D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int num_total)
{
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    int index = xIndex + N*yIndex;

    if (index < num_total)	{
        Output[index] = Input1[index] - Input2[index];
    }
}

__global__ void ROFcopy_kernel3D(float *Input, float* Output, int N, int M, int Z, int num_total)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        Output[index] = Input[index];
    }
}

__global__ void ROFResidCalc3D_kernel(float *Input1, float *Input2, float* Output, int N, int M, int Z, int num_total)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int k = blockDim.z * blockIdx.z + threadIdx.z;

    int index = (N*M)*k + i + N*j;

    if (index < num_total)	{
        Output[index] = Input1[index] - Input2[index];
    }
}


/////////////////////////////////////////////////
///////////////// HOST FUNCTION /////////////////
extern "C" int TV_ROF_GPU_main(float* Input, float* Output, float *infovector, float *lambdaPar, int lambda_is_arr, int iter, float tau, float epsil, int gpu_device, int N, int M, int Z)
{
     int deviceCount = -1; // number of devices
     hipGetDeviceCount(&deviceCount);
     if (deviceCount == 0) {
         fprintf(stderr, "No CUDA devices found\n");
          return -1;
      }
    checkCudaErrors(hipSetDevice(gpu_device));

    float re;
    re = 0.0f;
	  int ImSize, count, n;
	  count = 0; n = 0;
    float *d_input, *d_update, *d_D1, *d_D2, *d_update_prev=NULL, *lambdaPar_d=NULL;

	if (Z == 0) Z = 1;
	      ImSize = N*M*Z;
        CHECK(hipMalloc((void**)&d_input,ImSize*sizeof(float)));
        CHECK(hipMalloc((void**)&d_update,ImSize*sizeof(float)));
        CHECK(hipMalloc((void**)&d_D1,ImSize*sizeof(float)));
        CHECK(hipMalloc((void**)&d_D2,ImSize*sizeof(float)));
        if (epsil != 0.0f) checkCudaErrors( hipMalloc((void**)&d_update_prev,ImSize*sizeof(float)) );

        CHECK(hipMemcpy(d_input,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_update,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));

        /*dealing with spatially variant reglariser */
        if (lambda_is_arr == 1) {
          CHECK(hipMalloc((void**)&lambdaPar_d,ImSize*sizeof(float)));
          CHECK(hipMemcpy(lambdaPar_d,lambdaPar,ImSize*sizeof(float),hipMemcpyHostToDevice));
        }
        else {
          CHECK(hipMalloc((void**)&lambdaPar_d,1*sizeof(float)));
          CHECK(hipMemcpy(lambdaPar_d,lambdaPar,1*sizeof(float),hipMemcpyHostToDevice));
        }

        if (Z == 1) {
             // TV - 2D case
            dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
            dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));

            for(n=0; n < iter; n++) {

              if ((epsil != 0.0f) && (n % 5 == 0)) {
              ROFcopy_kernel2D<<<dimGrid,dimBlock>>>(d_update, d_update_prev, N, M, ImSize);
              checkCudaErrors( hipDeviceSynchronize() );
              checkCudaErrors(hipPeekAtLastError() );
              }
                /* calculate differences */
                D1_func2D<<<dimGrid,dimBlock>>>(d_update, d_D1, N, M);
                CHECK(hipDeviceSynchronize());
		            D2_func2D<<<dimGrid,dimBlock>>>(d_update, d_D2, N, M);
                CHECK(hipDeviceSynchronize());
                /*running main kernel*/
                TV_kernel2D<<<dimGrid,dimBlock>>>(d_D1, d_D2, d_update, d_input, lambdaPar_d, lambda_is_arr, tau, N, M);
                CHECK(hipDeviceSynchronize());

                if ((epsil != 0.0f) && (n % 5 == 0)) {
                /* calculate norm - stopping rules using the Thrust library */
                ROFResidCalc2D_kernel<<<dimGrid,dimBlock>>>(d_update, d_update_prev, d_D1, N, M, ImSize);
                checkCudaErrors( hipDeviceSynchronize() );
                checkCudaErrors( hipPeekAtLastError() );

                // setup arguments
		            square<float>        unary_op;
		            thrust::plus<float> binary_op;
                thrust::device_vector<float> d_vec(d_D1, d_D1 + ImSize);
		            float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
                thrust::device_vector<float> d_vec2(d_update, d_update + ImSize);
      		      float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

                // compute norm
                re = (reduction/reduction2);
                if (re < epsil)  count++;
                if (count > 3) break;
           	}

            }
        }
	 else {
	           // TV - 3D case
            dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
            dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKXSIZE));

            float *d_D3;
            CHECK(hipMalloc((void**)&d_D3,N*M*Z*sizeof(float)));

            for(n=0; n < iter; n++) {

              if ((epsil != 0.0f) && (n % 5 == 0)) {
              ROFcopy_kernel3D<<<dimGrid,dimBlock>>>(d_update, d_update_prev, N, M, Z, ImSize);
              checkCudaErrors( hipDeviceSynchronize() );
              checkCudaErrors(hipPeekAtLastError() );
              }
                /* calculate differences */
                D1_func3D<<<dimGrid,dimBlock>>>(d_update, d_D1, N, M, Z);
                CHECK(hipDeviceSynchronize());
		            D2_func3D<<<dimGrid,dimBlock>>>(d_update, d_D2, N, M, Z);
                CHECK(hipDeviceSynchronize());
                D3_func3D<<<dimGrid,dimBlock>>>(d_update, d_D3, N, M, Z);
                CHECK(hipDeviceSynchronize());
                /*running main kernel*/
                TV_kernel3D<<<dimGrid,dimBlock>>>(d_D1, d_D2, d_D3, d_update, d_input, lambdaPar_d, lambda_is_arr, tau, N, M, Z);
                CHECK(hipDeviceSynchronize());

                if ((epsil != 0.0f) && (n % 5 == 0)) {
                /* calculate norm - stopping rules using the Thrust library */
                ROFResidCalc3D_kernel<<<dimGrid,dimBlock>>>(d_update, d_update_prev, d_D1, N, M, Z, ImSize);
                checkCudaErrors( hipDeviceSynchronize() );
                checkCudaErrors( hipPeekAtLastError() );

                // setup arguments
                square<float>        unary_op;
                thrust::plus<float> binary_op;
                thrust::device_vector<float> d_vec(d_D1, d_D1 + ImSize);
                float reduction = std::sqrt(thrust::transform_reduce(d_vec.begin(), d_vec.end(), unary_op, 0.0f, binary_op));
                thrust::device_vector<float> d_vec2(d_update, d_update + ImSize);
                float reduction2 = std::sqrt(thrust::transform_reduce(d_vec2.begin(), d_vec2.end(), unary_op, 0.0f, binary_op));

                // compute norm
                re = (reduction/reduction2);
                if (re < epsil)  count++;
                if (count > 3) break;
              }
            }
            CHECK(hipFree(d_D3));
        }
        CHECK(hipMemcpy(Output,d_update,N*M*Z*sizeof(float),hipMemcpyDeviceToHost));
        if (epsil != 0.0f) hipFree(d_update_prev);
        CHECK(hipFree(d_input));
        CHECK(hipFree(d_update));
        CHECK(hipFree(d_D1));
        CHECK(hipFree(d_D2));
        CHECK(hipFree(lambdaPar_d));

	      infovector[0] = (float)(n);  /*iterations number (if stopped earlier based on tolerance)*/
        infovector[1] = re;  /* reached tolerance */
        checkCudaErrors( hipDeviceSynchronize() );
        return 0;
}
