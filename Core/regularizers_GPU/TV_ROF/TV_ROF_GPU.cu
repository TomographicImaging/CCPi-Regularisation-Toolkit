#include "hip/hip_runtime.h"
 /*
This work is part of the Core Imaging Library developed by
Visual Analytics and Imaging System Group of the Science Technology
Facilities Council, STFC

Copyright 2017 Daniil Kazantsev
Copyright 2017 Srikanth Nagella, Edoardo Pasca

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/ 

#include "TV_ROF_GPU.h"

/* C-OMP implementation of ROF-TV denoising/regularization model [1] (2D/3D case)
*
* Input Parameters:
* 1. Noisy image/volume [REQUIRED]
* 2. lambda - regularization parameter [REQUIRED]
* 3. tau - marching step for explicit scheme, ~0.1 is recommended [REQUIRED]
* 4. Number of iterations, for explicit scheme >= 150 is recommended [REQUIRED]
*
* Output:
* [1] Regularized image/volume

 * This function is based on the paper by
* [1] Rudin, Osher, Fatemi, "Nonlinear Total Variation based noise removal algorithms"
*
* D. Kazantsev, 2016-18
*/

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}
    
#define BLKXSIZE 8
#define BLKYSIZE 8
#define BLKZSIZE 8
    
#define BLKXSIZE2D 16
#define BLKYSIZE2D 16
#define EPS 1.0e-4
    
#define idivup(a, b) ( ((a)%(b) != 0) ? (a)/(b)+1 : (a)/(b) )

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__host__ __device__ int sign (float x)
{
        return (x > 0) - (x < 0);
}        
   
/*********************2D case****************************/    
    
    /* differences 1 */
    __global__ void D1_func2D(float* Input, float* D1, int N, int M)      
    {
		int i1, j1, i2, j2;
		float NOMx_1,NOMy_1,NOMy_0,denom1,denom2,T1;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + N*j;        
        
        if ((i >= 0) && (i < N) && (j >= 0) && (j < M)) {
            
            /* boundary conditions (Neumann reflections) */
                i1 = i + 1; if (i1 >= N) i1 = i-1;
                i2 = i - 1; if (i2 < 0) i2 = i+1;
                j1 = j + 1; if (j1 >= M) j1 = j-1;
                j2 = j - 1; if (j2 < 0) j2 = j+1; 
		
		     /* Forward-backward differences */
                NOMx_1 = Input[j1*N + i] - Input[index]; /* x+ */
                NOMy_1 = Input[j*N + i1] - Input[index]; /* y+ */                
                NOMy_0 = Input[index] - Input[j*N + i2]; /* y- */
                
                denom1 = NOMx_1*NOMx_1;
                denom2 = 0.5f*(sign((float)NOMy_1) + sign((float)NOMy_0))*(MIN(abs((float)NOMy_1),abs((float)NOMy_0)));
                denom2 = denom2*denom2;
                T1 = sqrt(denom1 + denom2 + EPS);
                D1[index] = NOMx_1/T1;	
		}		
	}       
    
    /* differences 2 */
    __global__ void D2_func2D(float* Input, float* D2, int N, int M)      
    {
		int i1, j1, i2, j2;
		float NOMx_1,NOMy_1,NOMx_0,denom1,denom2,T2;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + N*j;        
        
        if ((i >= 0) && (i < (N)) && (j >= 0) && (j < (M))) {
            
            /* boundary conditions (Neumann reflections) */
                i1 = i + 1; if (i1 >= N) i1 = i-1;
                i2 = i - 1; if (i2 < 0) i2 = i+1;
                j1 = j + 1; if (j1 >= M) j1 = j-1;
                j2 = j - 1; if (j2 < 0) j2 = j+1; 
		
                /* Forward-backward differences */
                NOMx_1 = Input[j1*N + i] - Input[index]; /* x+ */
                NOMy_1 = Input[j*N + i1] - Input[index]; /* y+ */
                NOMx_0 = Input[index] - Input[j2*N + i]; /* x- */
                
                denom1 = NOMy_1*NOMy_1;
                denom2 = 0.5f*(sign((float)NOMx_1) + sign((float)NOMx_0))*(MIN(abs((float)NOMx_1),abs((float)NOMx_0)));
                denom2 = denom2*denom2;
                T2 = sqrt(denom1 + denom2 + EPS);
                D2[index] = NOMy_1/T2;	
		}		
	}
    
    __global__ void TV_kernel2D(float *D1, float *D2, float *Update, float *Input, float lambda, float tau, int N, int M)    
    {
		int i2, j2;
		float dv1,dv2;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        
        int index = i + N*j;        
        
        if ((i >= 0) && (i < (N)) && (j >= 0) && (j < (M))) {
            
				/* boundary conditions (Neumann reflections) */                
                i2 = i - 1; if (i2 < 0) i2 = i+1;                
                j2 = j - 1; if (j2 < 0) j2 = j+1; 
                
				/* divergence components  */
                dv1 = D1[index] - D1[j2*N + i];
                dv2 = D2[index] - D2[j*N + i2];                                
                
                Update[index] =  Update[index] + tau*((dv1 + dv2) - lambda*(Update[index] - Input[index]));      
		
		}  
	}   
/*********************3D case****************************/    
 
    /* differences 1 */
    __global__ void D1_func3D(float* Input, float* D1, int dimX, int dimY, int dimZ)      
    {
		float NOMx_1, NOMy_1, NOMy_0, NOMz_1, NOMz_0, denom1, denom2,denom3, T1;
		int i1,i2,k1,j1,j2,k2;
		
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
      	int index = (dimX*dimY)*k + j*dimX+i;     
        
        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {
            
                    /* symmetric boundary conditions (Neuman) */
                    i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                    i2 = i - 1; if (i2 < 0) i2 = i+1;
                    j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                    j2 = j - 1; if (j2 < 0) j2 = j+1;
                    k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                    k2 = k - 1; if (k2 < 0) k2 = k+1;                    
                    
                    /* Forward-backward differences */
                    NOMx_1 = Input[(dimX*dimY)*k + j1*dimX + i] - Input[index]; /* x+ */
                    NOMy_1 = Input[(dimX*dimY)*k + j*dimX + i1] - Input[index]; /* y+ */                    
                    NOMy_0 = Input[index] - Input[(dimX*dimY)*k + j*dimX + i2]; /* y- */
                    
                    NOMz_1 = Input[(dimX*dimY)*k1 + j*dimX + i] - Input[index]; /* z+ */
                    NOMz_0 = Input[index] - Input[(dimX*dimY)*k2 + j*dimX + i]; /* z- */
                    
                    
                    denom1 = NOMx_1*NOMx_1;
                    denom2 = 0.5*(sign(NOMy_1) + sign(NOMy_0))*(MIN(abs(NOMy_1),abs(NOMy_0)));
                    denom2 = denom2*denom2;
                    denom3 = 0.5*(sign(NOMz_1) + sign(NOMz_0))*(MIN(abs(NOMz_1),abs(NOMz_0)));
                    denom3 = denom3*denom3;
                    T1 = sqrt(denom1 + denom2 + denom3 + EPS);
                    D1[index] = NOMx_1/T1;	
		}		
	}      

    /* differences 2 */
    __global__ void D2_func3D(float* Input, float* D2, int dimX, int dimY, int dimZ)      
    {
		float NOMx_1, NOMy_1, NOMx_0, NOMz_1, NOMz_0, denom1, denom2, denom3, T2;
		int i1,i2,k1,j1,j2,k2;
		
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
      	int index = (dimX*dimY)*k + j*dimX+i;     
        
        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {
                    /* symmetric boundary conditions (Neuman) */
                    i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                    i2 = i - 1; if (i2 < 0) i2 = i+1;
                    j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                    j2 = j - 1; if (j2 < 0) j2 = j+1;
                    k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                    k2 = k - 1; if (k2 < 0) k2 = k+1;
                    
                    
                    /* Forward-backward differences */
                    NOMx_1 = Input[(dimX*dimY)*k + (j1)*dimX + i] - Input[index]; /* x+ */
                    NOMy_1 = Input[(dimX*dimY)*k + (j)*dimX + i1] - Input[index]; /* y+ */
                    NOMx_0 = Input[index] - Input[(dimX*dimY)*k + (j2)*dimX + i]; /* x- */
                    NOMz_1 = Input[(dimX*dimY)*k1 + j*dimX + i] - Input[index]; /* z+ */
                    NOMz_0 = Input[index] - Input[(dimX*dimY)*k2 + (j)*dimX + i]; /* z- */
                    
                    
                    denom1 = NOMy_1*NOMy_1;
                    denom2 = 0.5*(sign(NOMx_1) + sign(NOMx_0))*(MIN(abs(NOMx_1),abs(NOMx_0)));
                    denom2 = denom2*denom2;
                    denom3 = 0.5*(sign(NOMz_1) + sign(NOMz_0))*(MIN(abs(NOMz_1),abs(NOMz_0)));
                    denom3 = denom3*denom3;
                    T2 = sqrt(denom1 + denom2 + denom3 + EPS);
                    D2[index] = NOMy_1/T2;
		}
	}
	
	  /* differences 3 */
    __global__ void D3_func3D(float* Input, float* D3, int dimX, int dimY, int dimZ)      
    {
		float NOMx_1, NOMy_1, NOMx_0, NOMy_0, NOMz_1, denom1, denom2, denom3, T3;
		int i1,i2,k1,j1,j2,k2;
		
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
      	int index = (dimX*dimY)*k + j*dimX+i;     
        
        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {

				i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                i2 = i - 1; if (i2 < 0) i2 = i+1;
                j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                j2 = j - 1; if (j2 < 0) j2 = j+1;
                k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                k2 = k - 1; if (k2 < 0) k2 = k+1;
                
                /* Forward-backward differences */
                NOMx_1 = Input[(dimX*dimY)*k + (j1)*dimX + i] - Input[index]; /* x+ */
                NOMy_1 = Input[(dimX*dimY)*k + (j)*dimX + i1] - Input[index]; /* y+ */
                NOMy_0 = Input[index] - Input[(dimX*dimY)*k + (j)*dimX + i2]; /* y- */
                NOMx_0 = Input[index] - Input[(dimX*dimY)*k + (j2)*dimX + i]; /* x- */
                NOMz_1 = Input[(dimX*dimY)*k1 + j*dimX + i] - Input[index]; /* z+ */
               
                denom1 = NOMz_1*NOMz_1;
                denom2 = 0.5*(sign(NOMx_1) + sign(NOMx_0))*(MIN(abs(NOMx_1),abs(NOMx_0)));
                denom2 = denom2*denom2;
                denom3 = 0.5*(sign(NOMy_1) + sign(NOMy_0))*(MIN(abs(NOMy_1),abs(NOMy_0)));
                denom3 = denom3*denom3;
                T3 = sqrt(denom1 + denom2 + denom3 + EPS);
                D3[index] = NOMz_1/T3;		
		}
	}

    __global__ void TV_kernel3D(float *D1, float *D2, float *D3, float *Update, float *Input, float lambda, float tau, int dimX, int dimY, int dimZ)    
    {
		float dv1, dv2, dv3;
		int i1,i2,k1,j1,j2,k2;
		int i = blockDim.x * blockIdx.x + threadIdx.x;
        int j = blockDim.y * blockIdx.y + threadIdx.y;
        int k = blockDim.z * blockIdx.z + threadIdx.z;
        
        int index = (dimX*dimY)*k + j*dimX+i;       
        
        if ((i >= 0) && (i < dimX) && (j >= 0) && (j < dimY) && (k >= 0) && (k < dimZ)) {
            
					/* symmetric boundary conditions (Neuman) */
                    i1 = i + 1; if (i1 >= dimX) i1 = i-1;
                    i2 = i - 1; if (i2 < 0) i2 = i+1;
                    j1 = j + 1; if (j1 >= dimY) j1 = j-1;
                    j2 = j - 1; if (j2 < 0) j2 = j+1;
                    k1 = k + 1; if (k1 >= dimZ) k1 = k-1;
                    k2 = k - 1; if (k2 < 0) k2 = k+1;
                    
                    /*divergence components */
                    dv1 = D1[index] - D1[(dimX*dimY)*k + j2*dimX+i];
                    dv2 = D2[index] - D2[(dimX*dimY)*k + j*dimX+i2];
                    dv3 = D3[index] - D3[(dimX*dimY)*k2 + j*dimX+i];
                    
                    Update[index] = Update[index] + tau*lambda*(dv1 + dv2 + dv3) + tau*(Update[index] - Input[index]);   
		
		}  
	}

/////////////////////////////////////////////////
// HOST FUNCTION
extern "C" void TV_ROF_GPU(float* Input, float* Output, int N, int M, int Z, int iter, float tau, float lambda)
{
	    // set up device
		int dev = 0;
		CHECK(hipSetDevice(dev));
		
        float *d_input, *d_update, *d_D1, *d_D2;
        
        CHECK(hipMalloc((void**)&d_input,N*M*Z*sizeof(float)));
        CHECK(hipMalloc((void**)&d_update,N*M*Z*sizeof(float)));
        CHECK(hipMalloc((void**)&d_D1,N*M*Z*sizeof(float)));
        CHECK(hipMalloc((void**)&d_D2,N*M*Z*sizeof(float)));
        
        CHECK(hipMemcpy(d_input,Input,N*M*Z*sizeof(float),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_update,Input,N*M*Z*sizeof(float),hipMemcpyHostToDevice));      
        
        if (Z > 1) {
			// TV - 3D case                 
            dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
            dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKXSIZE));            
            
            float *d_D3;
            CHECK(hipMalloc((void**)&d_D3,N*M*Z*sizeof(float)));
            
            for(int n=0; n < iter; n++) {
                /* calculate differences */
                D1_func3D<<<dimGrid,dimBlock>>>(d_update, d_D1, N, M, Z);				
                CHECK(hipDeviceSynchronize());
				D2_func3D<<<dimGrid,dimBlock>>>(d_update, d_D2, N, M, Z);				                
                CHECK(hipDeviceSynchronize());        
                D3_func3D<<<dimGrid,dimBlock>>>(d_update, d_D3, N, M, Z);				
                CHECK(hipDeviceSynchronize());        
                /*running main kernel*/
                TV_kernel3D<<<dimGrid,dimBlock>>>(d_D1, d_D2, d_D3, d_update, d_input, lambda, tau, N, M, Z);
                CHECK(hipDeviceSynchronize());
            }
            
            CHECK(hipFree(d_D3));         
        }
        else {
			// TV - 2D case
            dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
            dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));
             
            for(int n=0; n < iter; n++) {
                /* calculate differences */
                D1_func2D<<<dimGrid,dimBlock>>>(d_update, d_D1, N, M);				
                CHECK(hipDeviceSynchronize());
				D2_func2D<<<dimGrid,dimBlock>>>(d_update, d_D2, N, M);				
                CHECK(hipDeviceSynchronize());        
                /*running main kernel*/
                TV_kernel2D<<<dimGrid,dimBlock>>>(d_D1, d_D2, d_update, d_input, lambda, tau, N, M);
                CHECK(hipDeviceSynchronize());
            }
        }        
        CHECK(hipMemcpy(Output,d_update,N*M*Z*sizeof(float),hipMemcpyDeviceToHost));
        CHECK(hipFree(d_input));
        CHECK(hipFree(d_update));
        CHECK(hipFree(d_D1));
        CHECK(hipFree(d_D2));        
        hipDeviceReset(); 
}
